#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>
#include <cstdlib>

#define ARRAY_SIZE 1000

__global__ void scan(float* d_in, float* d_out){
	__shared__ float s[ARRAY_SIZE];
	int index = threadIdx.x;
	
	s[index] = d_in[index];
	__syncthreads();

	float local = 0;
	for(int add = 1; add < ARRAY_SIZE; add <<= 1){
		local = s[index];
		__syncthreads();

		if (add + index < ARRAY_SIZE)
			s[index + add] += local;	
		__syncthreads();
	}
	d_out[index] = s[index];
}



int main(void){

	float h_in[ARRAY_SIZE];
	float h_out[ARRAY_SIZE];

	for (int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float( rand() % 10 + 1 );
		//printf("%lf\n", h_in[i]);
	}
	
	const size_t BYTE_SIZE = ARRAY_SIZE *  sizeof(float);

	float* d_in;
	float* d_out;

	hipMalloc((void**)&d_in, BYTE_SIZE);
	hipMalloc((void**)&d_out, BYTE_SIZE);


	hipMemcpy(d_in, h_in, BYTE_SIZE, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	scan << <1, ARRAY_SIZE >> >(d_in, d_out);

	hipMemcpy(h_out, d_out, BYTE_SIZE, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < ARRAY_SIZE; i++){
		printf("%d, %lf: %lf\n", i, h_in[i], h_out[i]);
	}
	
	hipFree(d_in);
	hipFree(d_out);
	
	system("pause");
	return 0;
}