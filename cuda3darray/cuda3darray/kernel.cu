#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>


__global__ void matrixMultiply(hipPitchedPtr matrix1, hipExtent extent){
	//printf("matrixMultiply is called from: %d, %d", threadIdx.x, threadIdx.y);

	char* devPtr = (char*)matrix1.ptr;
	size_t pitch = matrix1.pitch;
	size_t slicePitch = pitch*extent.height;

	int x = threadIdx.x;
	int y = threadIdx.y;
	int z = threadIdx.z;

	char* slice = devPtr + z * slicePitch;
	float* row = (float*)(slice + y * pitch);
	printf("%d,%d,%d : %f\n", x, y, z, row[x]);

}

int main(void){
	static const size_t ROWNUM = 10;
	static const size_t COLNUM = 5;
	static const size_t Z = 2;

	float* h_data = new float[ROWNUM*COLNUM*Z];
	for (int i = 0; i < ROWNUM*COLNUM*Z; i++){
		h_data[i] = (float)i;
	}

	hipPitchedPtr h_dataPtr = make_hipPitchedPtr(h_data, ROWNUM*sizeof(float), ROWNUM, COLNUM);

	hipPitchedPtr d_matrix1Ptr;
	hipExtent extent = make_hipExtent(ROWNUM*sizeof(float), COLNUM, Z);

	hipMalloc3D(&d_matrix1Ptr, extent);

	printf("%d\n", d_matrix1Ptr.pitch);

	hipMemcpy3DParms params = { 0 };
	params.srcPtr = h_dataPtr;
	params.dstPtr = d_matrix1Ptr;
	params.extent = extent;
	params.kind = hipMemcpyHostToDevice;

	hipMemcpy3D(&params);
	printf("%s\n", hipGetErrorString(hipGetLastError()));


	dim3 dimen = dim3(ROWNUM, COLNUM, Z);
	matrixMultiply << <1, dimen >> >(d_matrix1Ptr, extent);
	
	delete[] h_data;

	hipFree(d_matrix1Ptr.ptr);
	system("pause");
	return 0;
}