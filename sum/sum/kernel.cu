#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>

#define ARRAY_SIZE 1000

__global__ void sum(float* d_in, float* d_sum){
	__shared__ float s[ARRAY_SIZE];
	int index = threadIdx.x;

	s[index] = d_in[index];
	__syncthreads();

	for (int i = 1; i < ARRAY_SIZE; i <<= 1){
		if (index%(i*2) == 0){
			s[index] += s[index + i];
		}
		__syncthreads();
	}
	if (index == 0){
		d_sum[0] = s[index];
	}
}


int main(void){
	const size_t BYTE_SIZE = ARRAY_SIZE *  sizeof(float);

	float h_in[ARRAY_SIZE];
	float h_sum[1];

	for (int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float(rand() % 10 + 1);
		//printf("%lf\n", h_in[i]);
	}

	float* d_in;
	float* d_sum;

	hipMalloc((void**)&d_in, BYTE_SIZE);
	hipMalloc((void**)&d_sum, sizeof(float));

	hipMemcpy(d_in, h_in, BYTE_SIZE, hipMemcpyHostToDevice);

	if(hipDeviceSynchronize() == hipSuccess)
		printf("memory copy is sucessful\n");

	sum << <1, ARRAY_SIZE >> >(d_in, d_sum);

	if (hipDeviceSynchronize() == hipSuccess)
		printf("sums is sucessful\n");

	hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

	printf("sum: %lf\n", h_sum[0]);
	
	hipFree(d_in);
	hipFree(d_sum);

	system("pause");
	return 0;
}