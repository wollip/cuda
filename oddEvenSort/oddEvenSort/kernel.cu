#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>
#include <cstdlib>
#include <math.h>

#define ARRAY_SIZE 1001

__device__ void swap(float* a, int index1, int index2){
	float temp = a[index1];
	a[index1] = a[index2];
	a[index2] = temp;
}

__global__ void oddEvenSort(float* d_in, float* d_out){
	__shared__ float s[ARRAY_SIZE];
	
	int index = 2*threadIdx.x;
	int swapIndex = index + 1;

	s[index] = d_in[index];
	s[swapIndex] = d_in[swapIndex];
	__syncthreads();
	
	bool even = true;
	for (int i = 0; i < ARRAY_SIZE; i++){
		if (swapIndex < ARRAY_SIZE && index >= 0){
			if (s[index] > s[swapIndex]){
				swap(s, index, swapIndex);
			}			
		}
		if (even){
			index++;
			swapIndex++;
			even = false;
		}
		else{
			index--;
			swapIndex--;
			even = true;
		}
		__syncthreads();
	}
	

	d_out[index] = s[2*threadIdx.x];
	d_out[swapIndex] = s[2*threadIdx.x + 1];
}

int main(void){
	float h_in[ARRAY_SIZE] , h_out[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = rand() % 10000;
	}

	const size_t BYTE_SIZE = ARRAY_SIZE*sizeof(float);

	float* d_in, *d_out;
	hipMalloc((void**)&d_in, BYTE_SIZE);
	hipMalloc((void**)&d_out, BYTE_SIZE);
	hipMemcpy(d_in, h_in, BYTE_SIZE, hipMemcpyHostToDevice);

	oddEvenSort << <1, ceil(ARRAY_SIZE/2) >> >(d_in, d_out);

	hipMemcpy(h_out, d_out, BYTE_SIZE, hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE; i++){
		printf("%d:%lf\n", i, h_out[i]);
	}
	
	hipFree(d_in);
	hipFree(d_out);

	system("pause");
	return 0;
}