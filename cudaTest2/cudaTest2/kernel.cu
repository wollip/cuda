#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <Windows.h>
#include <stdio.h>

#define NUM_BLOCKS 10
#define BLOCK_SIZE 1

__global__ void hello(){
	for (int i = 0; i < 10; i++){
		printf("this is block: %d, thread %d\n",i,  blockIdx.x);
		__syncthreads();
		printf("hi: %d, %d\n", i, blockIdx.x);
		__syncthreads();
	}

}

int main(void){
	hello<<<NUM_BLOCKS, BLOCK_SIZE>>>();
	
	printf("hello\n");
	hipDeviceSynchronize();

	printf("done\n");
	system("pause");
	return 0;
}